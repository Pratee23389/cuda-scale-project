
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void hello() {
  printf("Hello from CUDA kernel!\n");
}
int main() {
  hello<<<1,1>>>();
  hipDeviceSynchronize();
  std::cout<<"Host: done.\n";
  return 0;
}
